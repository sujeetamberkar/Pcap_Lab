#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string.h>

#define N 1024 // Maximum length of the sentence
#define MAX_WORD_LENGTH 64 // Assuming the word will not be longer than this

__constant__ char d_word[MAX_WORD_LENGTH];

__device__ bool isWordMatch(const char *sentence, int startIdx) {
    for (int i = 0; d_word[i] != '\0'; ++i) {
        if (sentence[startIdx + i] != d_word[i]) {
            return false;
        }
    }
    // Check if the next character is a space or end of the sentence, ensuring it's a complete word match
    char nextChar = sentence[startIdx + strlen(d_word)];
    return nextChar == ' ' || nextChar == '\0';
}

__global__ void countWordOccurrences(const char *sentence, int sentenceLength, unsigned int *count) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < sentenceLength) {
        // Check if current position is the start of a word (preceded by a space if not at the beginning) and matches the target word
        if ((i == 0 || sentence[i - 1] == ' ') && isWordMatch(sentence, i)) {
            atomicAdd(count, 1);
        }
    }
}

int main() {
    const char sentence[N] = "this is a test sentence with test as a test word";
    const char word[MAX_WORD_LENGTH] = "test"; // The word to find
    char *d_sentence;
    unsigned int count = 0, *d_count;

    int sentenceLength = strlen(sentence);

    hipMalloc((void**)&d_sentence, sentenceLength + 1);
    hipMalloc((void**)&d_count, sizeof(unsigned int));

    hipMemcpy(d_sentence, sentence, sentenceLength + 1, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(d_word), word, MAX_WORD_LENGTH); // Copy word to constant memory
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (sentenceLength + blockSize - 1) / blockSize;
    countWordOccurrences<<<numBlocks, blockSize>>>(d_sentence, sentenceLength, d_count);

    hipMemcpy(&count, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("The word '%s' occurs %u times in the sentence.\n", word, count);

    hipFree(d_sentence);
    hipFree(d_count);

    return 0;
}
