#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Declare device constant memory for the input string
__constant__ char d_str[64];

__global__ void Copy(char *rstr, int len) {
    int i = threadIdx.x;
    int startIdx = (len * (len + 1) / 2) - ((len - i) * (len - i + 1) / 2);

    for (int j = 0; j < len - i; j++) {
        rstr[startIdx + j] = d_str[j];
    }
}

int main() {
    char str[] = "PCAP";
    int len = strlen(str);
    int newsize =  (len*(len+1)) / 2; 
    char rstr[newsize]; // Ensure this is large enough to hold the result
    char *d_rstr;

    // Allocate device memory for the result string
    hipMalloc((void**)&d_rstr, 300 * sizeof(char));
    hipMemset(d_rstr, 0, 300 * sizeof(char)); // Initialize device memory to zeros

    // Copy the input string to constant memory on the device
    hipMemcpyToSymbol(HIP_SYMBOL(d_str), str, len * sizeof(char));

    // Launch the kernel
    Copy<<<1, len>>>(d_rstr, len);

    // Copy the result back to host memory
    hipMemcpy(rstr, d_rstr,newsize* sizeof(char),hipMemcpyDeviceToHost);
    rstr[newsize]='\0';
    printf("\nResultant String: %s\n", rstr);

    // Clean up
    hipFree(d_rstr);
    return 0;
}
