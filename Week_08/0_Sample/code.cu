#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define N 1024

__global__ void CUDACount(const char* A, unsigned int *d_count) {
    int i = threadIdx.x;
    if (A[i] == 'a')
        atomicAdd(d_count, 1);
}

int main() {
    char A[N];
    char *d_A;
    unsigned int count = 0, result; // Use non-pointer for single unsigned int values
    unsigned int *d_count;

    printf("Enter a string: ");
    fgets(A, N, stdin);
    A[strcspn(A, "\n")] = 0; // Replace newline with null terminator

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    hipMalloc((void**)&d_A, (strlen(A) + 1) * sizeof(char));
    hipMalloc((void**)&d_count, sizeof(unsigned int));

    hipMemcpy(d_A, A, (strlen(A) + 1) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_count, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

    CUDACount<<<1, strlen(A)>>>(d_A, d_count);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipMemcpy(&result, d_count, sizeof(unsigned int), hipMemcpyDeviceToHost);

    printf("Total occurrences of 'a': %u\n", result);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time Taken = %f ms\n", elapsedTime);

    hipFree(d_A);
    hipFree(d_count);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
