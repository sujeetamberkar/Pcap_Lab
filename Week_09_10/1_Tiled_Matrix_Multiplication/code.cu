// Write a program in CUDA to perform tiled matrix multiplication using 2D Grid and 2D Block
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <unistd.h>

#define BLOCK_WIDTH 2
#define TILE_WIDTH 2
#define WIDTH 4

__global__ void MatMulElementThreadShared(int *a,int *b,int *c)
{
    __shared__ int MDs[TILE_WIDTH][TILE_WIDTH];
    __shared__ int NDs[TILE_WIDTH][TILE_WIDTH];
    int m;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int row = by*TILE_WIDTH + ty;
    int col = bx*TILE_WIDTH + tx;

    int pVal = 0;
    for(m=0;m<WIDTH/TILE_WIDTH;m++)
    {
        MDs[ty][tx] = a[row*WIDTH + m*TILE_WIDTH + tx];
        NDs[ty][tx] = b[(m*TILE_WIDTH + ty)*WIDTH + col];
        __syncthreads();
        for(int k=0;k<TILE_WIDTH;k++)
        {
            pVal += MDs[ty][k]*NDs[k][tx];
        }
        __syncthreads();
    }
    c[row*WIDTH+col] = pVal;
}

int main() {
    int matA[WIDTH * WIDTH] = {1, 2, 3, 4,
                               5, 6, 7, 8,
                               9, 10, 11, 12,
                               13, 14, 15, 16};
    
    int matB[WIDTH * WIDTH] = {16, 15, 14, 13,
                               12, 11, 10, 9,
                               8, 7, 6, 5,
                               4, 3, 2, 1};
    int matProd[WIDTH * WIDTH];
    int *dA, *dB, *dC;

    hipMalloc(&dA, sizeof(int) * WIDTH * WIDTH);
    hipMalloc(&dB, sizeof(int) * WIDTH * WIDTH);
    hipMalloc(&dC, sizeof(int) * WIDTH * WIDTH);

    hipMemcpy(dA, matA, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);
    hipMemcpy(dB, matB, sizeof(int) * WIDTH * WIDTH, hipMemcpyHostToDevice);
    
    dim3 grid(WIDTH / BLOCK_WIDTH, WIDTH / BLOCK_WIDTH);
    dim3 block(BLOCK_WIDTH, BLOCK_WIDTH);

    MatMulElementThreadShared<<<grid, block>>>(dA, dB, dC);

    hipMemcpy(matProd, dC, sizeof(int) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

    printf("\n==Result==\n");
    for (int i = 0; i < WIDTH; i++) {
        for (int j = 0; j < WIDTH; j++) {
            printf("%d ", matProd[i * WIDTH + j]);
        }
        printf("\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    return 0;
}
