#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloFromGPU(void) {
    if (threadIdx.x == 10) {
        printf("Hello World from GPU!\n");
    }
}

int main(void) {
    // Launch a kernel on the GPU with one thread for each element.
    helloFromGPU<<<1, 256>>>();
    
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    
    return 0;
}
