#include "hip/hip_runtime.h"

#include <stdio.h> // Include for printf

__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}

int main(void) 
{
    int a, b, c; // host copies of variables a, b & c
    int *d_a, *d_b, *d_c; // device copies of variables a, b & c
    int size = sizeof(int);
    
    // Allocate space for device copies of a, b, c
    hipError_t err;
    err = hipMalloc((void **)&d_a, size);
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMalloc((void **)&d_b, size);
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    err = hipMalloc((void **)&d_c, size);
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }
    
    // Setup input values
    a = 3;
    b = 5;
    
    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
    
    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);
    
    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("Result: %d\n", c);
    
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}
