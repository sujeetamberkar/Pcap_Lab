#include <stdio.h>
// CUDA runtime
#include <hip/hip_runtime.h>

// Kernel definition
__global__ void vectorAdd(const int *A, const int *B, int *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(void) {
    int numElements = 100; // Example size of vectors
    size_t size = numElements * sizeof(int);
    int *h_A = (int *)malloc(size);
    int *h_B = (int *)malloc(size);
    int *h_C = (int *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // Allocate the device input vectors
    int *d_A = NULL;
    hipMalloc((void **)&d_A, size);
    int *d_B = NULL;
    hipMalloc((void **)&d_B, size);
    int *d_C = NULL;
    hipMalloc((void **)&d_C, size);

    // Copy the host input vectors to the device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector in host memory.
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the results
    for (int i = 0; i < numElements; ++i) {
        printf("%d + %d = %d\n", h_A[i], h_B[i], h_C[i]);
    }
    
    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}
