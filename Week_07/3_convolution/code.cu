#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void conv1D(int *N, int *M, int *P, int width, int mask_width) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k;
    int Pvalue = 0;
    int N_start_point = i - (mask_width / 2);
    for (k = 0; k < mask_width; k++) {
        if (N_start_point + k >= 0 && N_start_point + k < width) {
            Pvalue += N[N_start_point + k] * M[k];
        }
    }
    if (i < width) {
        P[i] = Pvalue;
    }
}

int main() {
    int width = 100;
    int mask_width = 7;
    int *N, *M, *P;
    int *d_N, *d_M, *d_P;

    // Allocate host memory
    N = (int *)malloc(width * sizeof(int));
    M = (int *)malloc(mask_width * sizeof(int));
    P = (int *)malloc(width * sizeof(int));

    // Initialize host arrays
    for (int i = 0; i < width; i++) {
        N[i] = i; // Example: Initialize all elements to 1 for simplicity
    }
    for (int i = 0; i < mask_width; i++) {
        M[i] = i; // Example: Initialize all mask elements to 1 for simplicity
    }

    // Allocate device memory
    hipMalloc(&d_N, width * sizeof(int));
    hipMalloc(&d_M, mask_width * sizeof(int));
    hipMalloc(&d_P, width * sizeof(int));

    // Copy from host to device
    hipMemcpy(d_N, N, width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_M, M, mask_width * sizeof(int), hipMemcpyHostToDevice);

    // Define block size and number of blocks
    int threadsPerBlock = 256;
    int blocksPerGrid = (width + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    conv1D<<<blocksPerGrid, threadsPerBlock>>>(d_N, d_M, d_P, width, mask_width);

    // Copy result back to host
    hipMemcpy(P, d_P, width * sizeof(int), hipMemcpyDeviceToHost);


    // Print N
    printf("N = ");
    for (int i = 0; i < width; i++) {
        printf("%d ", N[i]);
    }
    printf("\n");

    // Print M
    printf("M = ");
    for (int i = 0; i < mask_width; i++) {
        printf("%d ", M[i]);
    }
    printf("\n");

    // Print P
    printf("P = ");
    for (int i = 0; i < width; i++) {
        printf("%d ", P[i]);
    }
    printf("\n");


    // Free device memory
    hipFree(d_N);
    hipFree(d_M);
    hipFree(d_P);

    // Free host memory
    free(N);
    free(M);
    free(P);

    return 0;
}
