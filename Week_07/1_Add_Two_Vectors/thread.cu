#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int index = threadIdx.x; // Since we're using N threads in a single block
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}

int main() {
    int N = 100; // Now N is 100
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    // Allocate memory on the host
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Allocate memory on the device
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i * 2;
    }

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch the vectorAdd kernel on 1 block with N threads
    vectorAdd<<<1, N>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    // Display the result
    for (int i = 0; i < N; i++) {
        printf("%d  +%d = %d\n",a[i],b[i],c[i]);
    }
    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);


    return 0;
}
