#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(int *a, int *b, int *c, int N) {
    int i = threadIdx.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    int N = 1024; // Example value; ensure N <= 1024 for most GPUs
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);

    // Allocate memory on the host
    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(size);

    // Allocate memory on the device
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Initialize input vectors
    for (int i = 0; i < N; i++) {
        a[i] = i;
        b[i] = i*2;
    }

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch the vectorAdd kernel
    vectorAdd<<<1, N>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Display the result
    for (int i = 0; i < N; i++) {
        printf("%d  +%d = %d\n",a[i],b[i],c[i]);
    }

    // Cleanup
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
