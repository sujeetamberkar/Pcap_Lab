#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define PI 3.14159265358979323846

// CUDA kernel to compute the sine of angles in radians
__global__ void computeSine(const float *anglesRadians, float *sineValues, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        sineValues[index] = sinf(anglesRadians[index]);
    }
}

int main() {
    int n = 1000;
    float *anglesRadians, *sineValues;
    float *d_anglesRadians, *d_sineValues;

    // Allocate host memory
    anglesRadians = (float *)malloc(n * sizeof(float));
    sineValues = (float *)malloc(n * sizeof(float));

    // Initialize angles with random values in radians
    srand(time(NULL)); // Seed the random number generator
    for (int i = 0; i < n; i++) {
        anglesRadians[i] = ((float)rand() / RAND_MAX) * 2 * PI; // Random angle in radians
    }

    // Allocate device memory
    hipMalloc(&d_anglesRadians, n * sizeof(float));
    hipMalloc(&d_sineValues, n * sizeof(float));

    // Copy angles to device
    hipMemcpy(d_anglesRadians, anglesRadians, n * sizeof(float), hipMemcpyHostToDevice);

    // Launch the kernel
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    computeSine<<<numBlocks, blockSize>>>(d_anglesRadians, d_sineValues, n);

    // Copy results back to host
    hipMemcpy(sineValues, d_sineValues, n * sizeof(float), hipMemcpyDeviceToHost);

    // Print angles in radians and their sine values
    printf("Angle (Radians)      sine(Angle)\n");
    for (int i = 0; i < n; i++) {
        printf("%f      %f\n", anglesRadians[i], sineValues[i]);
    }

    // Cleanup
    hipFree(d_anglesRadians);
    hipFree(d_sineValues);
    free(anglesRadians);
    free(sineValues);

    return 0;
}
